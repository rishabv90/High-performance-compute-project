#include "hip/hip_runtime.h"
#define _CRT_SECURE_NO_WARNINGS
#include <string>
#include <stdio.h>
#include <wb.h>
#include "RGBtoYUV.cpp"
#include "Greyscale.cpp"
#include "colorInvariant.cpp"
#include "Erosion.cpp"
#include "MaskGeneration.cpp"
#include "Smooth.cpp"
#include "sumProc5.cpp"
#include "map1Proc5.cpp"
#include "proc5.cpp"
#include "map2Proc5.cpp"


#define NUM_BINS 256
#define SMOOTH_KERNEL_VERSION 0	// Define Smooth Version 0 = 2D Shared Memory, 1 = 1D Shared Memory, 2 = 2D Global Memory

#define USE_STREAMING
#define USE_STREAM_EVENTS

//Canonical way to check for errors in CUDA - https://stackoverflow.com/a/14038590
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t errCode, const char *file, int line, bool abort=true) {
  if (errCode != hipSuccess) {
    fprintf(stderr, "GPU Assertion: %s %s %d\n", hipGetErrorString(errCode), file, line);
    if (abort) exit(errCode);
  }
}

static const int erosionStrelWidth = 5, erosionStrelHeight = 5;
static const float hostErosionStrel[erosionStrelHeight][erosionStrelWidth] = {
  {1, 1, 1, 1, 1},
  {1, 1, 1, 1, 1},
  {1, 1, 1, 1, 1},
  {1, 1, 1, 1, 1},
  {1, 1, 1, 1, 1}
};

static const int kernelWidth = 5, kernelHeight = 5;
static const float hostKernelData[kernelWidth][kernelHeight] = {
  {0.04, 0.04, 0.04, 0.04, 0.04},
  {0.04, 0.04, 0.04, 0.04, 0.04},
  {0.04, 0.04, 0.04, 0.04, 0.04},
  {0.04, 0.04, 0.04, 0.04, 0.04},
  {0.04, 0.04, 0.04, 0.04, 0.04}
};

void sumProc5Host(float* sum_input, unsigned int size, float* totalSum){
	// Set up number of threads and blocks
	unsigned int blockSize = 1024; 
	unsigned int maxElementsPerBlock = blockSize * 2; // due to binary tree nature of algorithm

	unsigned int gridSize = 0;
	if (size <= maxElementsPerBlock)
	{
		gridSize = (unsigned int)std::ceil(float(size) / float(maxElementsPerBlock));
	}
	else
	{
		gridSize = size / maxElementsPerBlock;
		if (size % maxElementsPerBlock != 0)
			gridSize++;
	}

	// Allocate memory for array of total sums produced by each block
	// Array length must be the same as number of blocks / grid size
	float* deviceBlockSums;
	gpuErrchk(hipMalloc(&deviceBlockSums, sizeof(float) * gridSize));
	gpuErrchk(hipMemset(deviceBlockSums, 0, sizeof(float) * gridSize));

	// Sum data allocated for each block
	sumProc5<<<gridSize, blockSize, sizeof(float) * maxElementsPerBlock>>>(deviceBlockSums, sum_input, size);

	// Sum each block's total sums (to get global total sum)
	// Use basic implementation if number of total sums is <= 2048
	// Else, recurse on this same function
	if (gridSize <= maxElementsPerBlock)
	{
		gpuErrchk(hipMemset(totalSum, 0, sizeof(float)));
		sumProc5<<<1, blockSize, sizeof(float) * maxElementsPerBlock>>>(totalSum, deviceBlockSums, gridSize);
	}
	else
	{
		float* sum_input_block_sums;
		gpuErrchk(hipMalloc(&sum_input_block_sums, sizeof(float) * gridSize));
		gpuErrchk(hipMemcpy(sum_input_block_sums, deviceBlockSums, sizeof(float) * gridSize, hipMemcpyDeviceToDevice));
		sumProc5Host(sum_input_block_sums, gridSize, totalSum);
		gpuErrchk(hipFree(sum_input_block_sums));
	}

	gpuErrchk(hipFree(deviceBlockSums));
}

//-------------------------------------------------
//---------------- MAIN FUNCTION:
	// Arguments:
	// ./shadow_removal <- This program call
	// input_image      <- The input image to have shadow removed
	// output_directory <- The directory to produce output images in
	// kernel_file      <- The convolution kernel file

int main(int argc, char** argv) {
  if (argc != 3) {
	fprintf(stderr, "Argument count is: %d and needs to be 3\n", argc);
    fprintf(stderr, "Usage: ./shadow_removal input_image output_directory\n");
    fprintf(stderr, "/(-_-)\\\n");
    return 1;
  }

  //----------------------------------------------------------------------------------------//
  //-------------------------------- Host Variables ----------------------------------------//
  //----------------------------------------------------------------------------------------//
  //--------- Process 0: RGB Image Input
  int imageChannels;
  int imageWidth;
  int imageHeight;
  
  char *inputImageFile = argv[1];
  std::string baseOutputDir(argv[2]);

  if (baseOutputDir.back() != '/') {
    baseOutputDir.append("/");
  }

  wbImage_t inputImage = wbImport(inputImageFile);
  imageWidth = wbImage_getWidth(inputImage);
  imageHeight = wbImage_getHeight(inputImage);
  imageChannels = wbImage_getChannels(inputImage);
  float *hostInputImageData = wbImage_getData(inputImage);

  // -------- Setting up streaming and events for synchronization
#ifdef USE_STREAMING
  hipStream_t colorspaceStream, yuvStream, grayscaleStream, resultStream;
  hipStreamCreate(&colorspaceStream);
  hipStreamCreate(&yuvStream);
  hipStreamCreate(&grayscaleStream);
  hipStreamCreate(&resultStream);
#endif

#if defined(USE_STREAM_EVENTS) && defined(USE_STREAMING)
  hipEvent_t colorspaceCompleteEvent, yuvCompleteEvent, grayscaleCompleteEvent;
  hipEventCreate(&colorspaceCompleteEvent);
  hipEventCreate(&yuvCompleteEvent);
  hipEventCreate(&grayscaleCompleteEvent);
#endif

  //--------- Process 1: YUV Conversion
  float *hostOutputImageDataYUV = (float*) malloc(imageWidth*imageHeight*imageChannels*sizeof(float));

  //--------- Process 1: Color Invariance
  float *hostOutputImageDataColorInvariant = (float*) malloc(imageWidth*imageHeight*imageChannels*sizeof(float));

  //--------- Process 1: Greyscale Conversion
  float *hostOutputImageDataGreyScale = (float*) malloc(imageWidth*imageHeight*sizeof(float));

  //--------- Process 2: YUV Masking
  float *hostOutputImageDataCbMask = (float*) malloc(imageWidth*imageHeight*sizeof(float));

  //--------- Process 2: Greyscale Masking
  float *hostOutputImageDataGrayMask = (float*) malloc(imageWidth*imageHeight*sizeof(float));

  //--------- Process 3: Smoothing
  float *hostOutputImageDataSmooth = (float*) malloc(imageWidth*imageHeight*sizeof(float));

  //--------- Process 4: Light Mask Erosion
  float *hostOutputImageErodedLight = (float*) malloc(imageWidth*imageHeight*sizeof(float));

  //--------- Process 4: Shadow Mask Erosion
  float *hostOutputImageErodedShadow = (float*) malloc(imageWidth*imageHeight*sizeof(float));

  //--------- Process 5: Ratio & Final Image
  float *hostResultImageData = (float*) malloc(imageWidth*imageHeight*imageChannels*sizeof(float));

  //------------------------------------------------------------------------------------------//
  //-------------------------------- Device Variables ----------------------------------------//
  //------------------------------------------------------------------------------------------//
  //--------- Process 0: RGB Image Input
  float *deviceInputImageData;
  gpuErrchk(hipMalloc((void **) &deviceInputImageData, imageWidth * imageHeight * imageChannels * sizeof(float)));

  //--------- Process 1: YUV Conversion
  float *deviceYUVOutputImageData;
  gpuErrchk(hipMalloc((void **) &deviceYUVOutputImageData, imageWidth * imageHeight * imageChannels * sizeof(float)));

  //--------- Process 1: Color Invariance
  float *deviceColorInvariantOutputImageData;
  gpuErrchk(hipMalloc((void **)&deviceColorInvariantOutputImageData, imageWidth * imageHeight * sizeof(float) * imageChannels));

  //--------- Process 1: Greyscale Conversion
  float *deviceGreyscaleOutputImageData;
  float *redData;
  float *greenData;
  float *blueData;
  gpuErrchk(hipMalloc((void **)&deviceGreyscaleOutputImageData, imageWidth * imageHeight * sizeof(float)));
  
  //For coalesced accesses after first process
  gpuErrchk(hipMalloc((void **)&redData, imageWidth * imageHeight * sizeof(float)));
  gpuErrchk(hipMalloc((void **)&greenData, imageWidth * imageHeight * sizeof(float)));
  gpuErrchk(hipMalloc((void **)&blueData, imageWidth * imageHeight * sizeof(float)));

  //--------- Process 2: YUV Masking
  float *deviceCBMaskOutputImageData;
  unsigned int *deviceCbBins;
  unsigned int *deviceCbHistogramSum;
  unsigned int *deviceGreyHistogramSum;
  float *deviceGrayOmega;
  float *deviceCbOmega;
  float *deviceGrayMu;
  float *deviceCbMu;
  float *deviceGraySigmaBSquared;
  float *deviceCbSigmaBSquared;
  float *deviceGrayThreshold;
  float *deviceCbThreshold;
  
  gpuErrchk(hipMalloc((void **)&deviceCBMaskOutputImageData, imageWidth * imageHeight * sizeof(float)));
  gpuErrchk(hipMalloc((void **)&deviceCbBins, NUM_BINS * sizeof(int)));
  gpuErrchk(hipMalloc((void **)&deviceCbHistogramSum, sizeof(int)));
  gpuErrchk(hipMalloc((void **)&deviceGreyHistogramSum, sizeof(int)));
  gpuErrchk(hipMalloc((void **)&deviceGrayOmega, NUM_BINS * sizeof(float)));
  gpuErrchk(hipMalloc((void **)&deviceCbOmega, NUM_BINS * sizeof(float)));
  gpuErrchk(hipMalloc((void **)&deviceGrayMu, NUM_BINS * sizeof(float)));
  gpuErrchk(hipMalloc((void **)&deviceCbMu, NUM_BINS * sizeof(float)));
  gpuErrchk(hipMalloc((void **)&deviceGraySigmaBSquared, NUM_BINS * sizeof(float)));
  gpuErrchk(hipMalloc((void **)&deviceCbSigmaBSquared, NUM_BINS * sizeof(float)));
  gpuErrchk(hipMalloc((void **)&deviceGrayThreshold, sizeof(float)));
  gpuErrchk(hipMalloc((void **)&deviceCbThreshold, sizeof(float)));

  //--------- Process 2: Greyscale Masking
  float *deviceGreyMaskOutputImageData;
  unsigned int *deviceGreyBins;

  gpuErrchk(hipMalloc((void **)&deviceGreyMaskOutputImageData, imageWidth * imageHeight * sizeof(float))); 
  gpuErrchk(hipMalloc((void **)&deviceGreyBins, NUM_BINS * sizeof(int)));

  //--------- Process 3: Smoothing
  float *deviceSmoothOutputImageData;
  float *deviceMaskData;

  gpuErrchk(hipMalloc((void **)&deviceSmoothOutputImageData, imageWidth * imageHeight * sizeof(float)));
  gpuErrchk(hipMalloc((void **)&deviceMaskData, kernelWidth * kernelHeight * sizeof(float)));

  //--------- Process 4: Light Mask Erosion
  float *deviceErodedLight;
  float *deviceStrel;

  gpuErrchk(hipMalloc((void **)&deviceErodedLight, imageHeight * imageWidth * sizeof(float)));
  gpuErrchk(hipMalloc((void **)&deviceStrel, erosionStrelHeight * erosionStrelWidth * sizeof(float)));

  //--------- Process 4: Shadow Mask Erosion
  float *deviceErodedShadow;

  gpuErrchk(hipMalloc((void **)&deviceErodedShadow, imageHeight * imageWidth * sizeof(float)));

  //--------- Process 5: Final Image
  float *deviceShadowRedArray;
  float *deviceShadowGreenArray;
  float *deviceShadowBlueArray;
  float *deviceLightRedArray;
  float *deviceLightGreenArray;
  float *deviceLightBlueArray;
  //Sums
  float *deviceErodedShadowSum;
  float *deviceErodedLightSum;
  float *deviceShadowRedArraySum;
  float *deviceShadowGreenArraySum;
  float *deviceShadowBlueArraySum;
  float *deviceLightRedArraySum;
  float *deviceLightGreenArraySum;
  float *deviceLightBlueArraySum;
  //
  float *deviceResultImageData;
  
  gpuErrchk(hipMalloc((void **)&deviceShadowRedArray, imageHeight * imageWidth * sizeof(float)));
  gpuErrchk(hipMalloc((void **)&deviceShadowGreenArray, imageHeight * imageWidth * sizeof(float)));
  gpuErrchk(hipMalloc((void **)&deviceShadowBlueArray, imageHeight * imageWidth * sizeof(float)));
  gpuErrchk(hipMalloc((void **)&deviceLightRedArray, imageHeight * imageWidth * sizeof(float)));
  gpuErrchk(hipMalloc((void **)&deviceLightGreenArray, imageHeight * imageWidth * sizeof(float)));
  gpuErrchk(hipMalloc((void **)&deviceLightBlueArray, imageHeight * imageWidth * sizeof(float)));

  gpuErrchk(hipMalloc((void **)&deviceShadowRedArraySum, sizeof(float)));
  gpuErrchk(hipMalloc((void **)&deviceShadowGreenArraySum, sizeof(float)));
  gpuErrchk(hipMalloc((void **)&deviceShadowBlueArraySum, sizeof(float)));
  gpuErrchk(hipMalloc((void **)&deviceLightRedArraySum, sizeof(float)));
  gpuErrchk(hipMalloc((void **)&deviceLightGreenArraySum, sizeof(float)));
  gpuErrchk(hipMalloc((void **)&deviceLightBlueArraySum,  sizeof(float)));
  gpuErrchk(hipMalloc((void **)&deviceErodedShadowSum,  sizeof(float)));
  gpuErrchk(hipMalloc((void **)&deviceErodedLightSum,  sizeof(float)));
  
  gpuErrchk(hipMalloc((void **)&deviceResultImageData, imageHeight * imageWidth * imageChannels * sizeof(float)));


  //-----------------------------------------------------------------------------------------------------//
  //-------------------------------- Data Transfer (Host -> GPU) ----------------------------------------//
  //-----------------------------------------------------------------------------------------------------//
  gpuErrchk(hipMemcpy(deviceInputImageData, hostInputImageData,
             imageWidth * imageHeight * imageChannels * sizeof(float),
             hipMemcpyHostToDevice));	 
			 
  #ifndef USE_STREAMING
  gpuErrchk(hipMemcpy(deviceMaskData, hostKernelData,
             kernelWidth * kernelHeight * sizeof(float), 
			       hipMemcpyHostToDevice));

  gpuErrchk(hipMemcpy(deviceStrel, hostErosionStrel,
             erosionStrelWidth * erosionStrelHeight * sizeof(float),
             hipMemcpyHostToDevice));
  #else
  gpuErrchk(hipMemcpyAsync(deviceMaskData, hostKernelData,
             kernelWidth * kernelHeight * sizeof(float),
             hipMemcpyHostToDevice, yuvStream));

  gpuErrchk(hipMemcpyAsync(deviceStrel, hostErosionStrel,
             erosionStrelWidth * erosionStrelHeight * sizeof(float),
             hipMemcpyHostToDevice, grayscaleStream));
  #endif


  //-------------------------------------------------------------------------------------------------//
  //-------------------------------- STOP! IT'S KERNEL TIME! ----------------------------------------//
  //-------------------------------------------------------------------------------------------------//

  // Shared grid or block dimension tuples
  dim3 dimGridHisto(2, 1, 1);
  dim3 dimBlockHisto(1024, 1, 1);

  dim3 dimGridCumSum(1, 1, 1);
  dim3 dimBlockCumSum(NUM_BINS, 1, 1);

  dim3 dimGridMasking((imageWidth - 1) / 16 + 1, (imageHeight - 1)/16 + 1, 1);
  dim3 dimBlockMasking(16, 16, 1);

  dim3 dimGridErosion((imageWidth - 1) / 16 + 1, (imageHeight - 1)/16 + 1, 1);
  dim3 dimBlockErosion(16, 16, 1);

  // Kernel Launches:
  // Color Invariant -> Grayscale processing, and YUV
  //--------- Process 1: Color Invariance
  dim3 dimGridColorInvariance((imageWidth - 1) / 16 + 1, (imageHeight - 1)/16 + 1, 1);
  dim3 dimBlockColorInvariance(16, 16, 1);

  hipEvent_t astartEvent, astopEvent;
  float aelapsedTime;
  hipEventCreate(&astartEvent);
  hipEventCreate(&astopEvent);
  hipEventRecord(astartEvent, 0);

#ifdef USE_STREAMING
  YUVandCItoGray<<<dimGridColorInvariance, dimBlockColorInvariance, 0, colorspaceStream>>>(deviceGreyscaleOutputImageData, deviceYUVOutputImageData, deviceInputImageData, redData, greenData, blueData, imageWidth, imageHeight); //implemented
#else
  YUVandCItoGray<<<dimGridColorInvariance, dimBlockColorInvariance>>>(deviceGreyscaleOutputImageData, deviceYUVOutputImageData, deviceInputImageData, redData, greenData, blueData, imageWidth, imageHeight);



#endif
hipDeviceSynchronize();
hipEventRecord(astopEvent, 0);
hipEventSynchronize(astopEvent);
hipEventElapsedTime(&aelapsedTime, astartEvent, astopEvent);
printf("Total execution time (ms) %f for YUVandCItoGray \n",aelapsedTime);
  // Wait until this stream is done as all other streams depend on the colorspace transform
#if defined(USE_STREAM_EVENTS) && defined(USE_STREAMING)
  hipEventRecord(colorspaceCompleteEvent, colorspaceStream);
  hipStreamWaitEvent(grayscaleStream, colorspaceCompleteEvent, 0);
#elif defined(USE_STREAMING)
  hipStreamSynchronize(colorspaceStream);
#endif
  
  //--------- Process 1: Greyscale Conversion, done previously


hipEvent_t event1StartEvent, event1StopEvent;
float event1ElapsedTime;
hipEventCreate(&event1StartEvent);
hipEventCreate(&event1StopEvent);
hipEventRecord(event1StartEvent, 0);
  //--------- Process 2: Greyscale Masking

#ifdef USE_STREAMING
  histogramKernel<<<dimGridHisto, dimBlockHisto, 0, grayscaleStream>>>(deviceGreyscaleOutputImageData, deviceGreyBins, imageWidth * imageHeight, true); //implemented
#else
  histogramKernel<<<dimGridHisto, dimBlockHisto>>>(deviceGreyscaleOutputImageData, deviceGreyBins, imageWidth * imageHeight, true); 
#endif

histogramSumKernel<<<dimGridCumSum, dimBlockCumSum>>>(deviceGreyBins, deviceGreyHistogramSum);

  //--------- Process 2: Greyscale Masking
#ifdef USE_STREAMING
  cumSumOne<<<dimGridCumSum, dimBlockCumSum, 0, grayscaleStream>>>(deviceGreyBins, deviceGrayOmega, imageWidth * imageHeight, deviceGreyHistogramSum); //pending
#else
  cumSumOne<<<dimGridCumSum, dimBlockCumSum>>>(deviceGreyBins, deviceGrayOmega, imageWidth * imageHeight, deviceGreyHistogramSum);
#endif
  
  //--------- Process 2: Greyscale Masking
#ifdef USE_STREAMING
  cumSumTwo<<<dimGridCumSum, dimBlockCumSum, 0, grayscaleStream>>>(deviceGreyBins, deviceGrayMu, imageWidth * imageHeight, deviceGreyHistogramSum); //pending
#else
  cumSumTwo<<<dimGridCumSum, dimBlockCumSum>>>(deviceGreyBins, deviceGrayMu, imageWidth * imageHeight, deviceGreyHistogramSum);
#endif
  
  //--------- Process 2: Greyscale Masking
#ifdef USE_STREAMING
  compSigmaBSquared<<<dimGridCumSum, dimBlockCumSum, 0, grayscaleStream>>>(deviceGraySigmaBSquared, deviceGrayOmega, deviceGrayMu);
#else
  compSigmaBSquared<<<dimGridCumSum, dimBlockCumSum>>>(deviceGraySigmaBSquared, deviceGrayOmega, deviceGrayMu);
#endif
  //--------- Process 2: Greyscale Masking
#ifdef USE_STREAMING
  argmax<<<1, 256, 0, grayscaleStream>>>(deviceGrayThreshold, deviceGraySigmaBSquared); //pending
#else
  argmax<<<1, 256>>>(deviceGrayThreshold, deviceGraySigmaBSquared);
#endif
  
  //--------- Process 2: Greyscale Masking
#ifdef USE_STREAMING
  maskGeneration<<<dimGridMasking, dimBlockMasking, 0, grayscaleStream>>>(deviceGreyscaleOutputImageData, deviceGreyMaskOutputImageData, deviceGrayThreshold, imageWidth, imageHeight, 1, true);
#else
  maskGeneration<<<dimGridMasking, dimBlockMasking>>>(deviceGreyscaleOutputImageData, deviceGreyMaskOutputImageData, deviceGrayThreshold, imageWidth, imageHeight, 1, true);
#endif
hipDeviceSynchronize();
hipEventRecord(event1StopEvent, 0);
hipEventSynchronize(event1StopEvent);
hipEventElapsedTime(&event1ElapsedTime, event1StartEvent, event1StopEvent);
printf("Total execution time (ms) %f for Greyscale Mask Generation \n",event1ElapsedTime);

hipEvent_t event2StartEvent, event2StopEvent;
float event2ElapsedTime;
hipEventCreate(&event2StartEvent);
hipEventCreate(&event2StopEvent);
hipEventRecord(event2StartEvent, 0);
  //--------- Process 4: Light Mask Erosion
#ifdef USE_STREAMING
  maskErosion<<<dimGridErosion, dimBlockErosion, 0, grayscaleStream>>>(deviceErodedLight, deviceGreyMaskOutputImageData, deviceStrel, imageWidth, imageHeight, true);
#else
  maskErosion<<<dimGridErosion, dimBlockErosion>>>(deviceErodedLight, deviceGreyMaskOutputImageData, deviceStrelLight, imageWidth, imageHeight, true);
#endif
hipDeviceSynchronize();
hipEventRecord(event2StopEvent, 0);
hipEventSynchronize(event2StopEvent);
hipEventElapsedTime(&event2ElapsedTime, event2StartEvent, event2StopEvent);
printf("Total execution time (ms) %f for light erosion \n",event2ElapsedTime);
hipEvent_t event6StartEvent, event6StopEvent;
float event6ElapsedTime;
hipEventCreate(&event6StartEvent);
hipEventCreate(&event6StopEvent);
hipEventRecord(event6StartEvent, 0);
  //--------- Process 4: Shadow Mask Erosion
#ifdef USE_STREAMING  
  maskErosion<<<dimGridErosion, dimBlockErosion, 0, grayscaleStream>>>(deviceErodedShadow, deviceGreyMaskOutputImageData, deviceStrel, imageWidth, imageHeight, false);
#else
  maskErosion<<<dimGridErosion, dimBlockErosion>>>(deviceErodedShadow, deviceGreyMaskOutputImageData, deviceStrelShadow, imageWidth, imageHeight, false);
#endif
hipDeviceSynchronize();
  hipEventRecord(event6StopEvent, 0);
  hipEventSynchronize(event6StopEvent);
  hipEventElapsedTime(&event6ElapsedTime, event6StartEvent, event6StopEvent);
  printf("Total execution time (ms) %f for shadow mask erosion \n",event6ElapsedTime);

#if defined(USE_STREAM_EVENTS) && defined(USE_STREAMING)
  hipEventRecord(grayscaleCompleteEvent, grayscaleStream);
  hipStreamWaitEvent(yuvStream, colorspaceCompleteEvent, 0);
#endif

  // YUV processing
  //--------- Process 1: YUV Conversion, done previously
  dim3 dimGridYUVConversion((imageWidth - 1) / 16 + 1, (imageHeight - 1)/16 + 1, 1);
  dim3 dimBlockYUVConversion(16, 16, 1);

  hipEvent_t event3StartEvent, event3StopEvent;
  float event3ElapsedTime;
  hipEventCreate(&event3StartEvent);
  hipEventCreate(&event3StopEvent);
  hipEventRecord(event3StartEvent, 0);
  //--------- Process 2: YUV Masking
#ifdef USE_STREAMING
  histogramKernel<<<dimGridHisto, dimBlockHisto, 0, yuvStream>>>(deviceYUVOutputImageData, deviceCbBins, imageWidth * imageHeight, false); //implemented
#else
  histogramKernel<<<dimGridHisto, dimBlockHisto>>>(deviceYUVOutputImageData, deviceCbBins, imageWidth * imageHeight, false);
#endif

histogramSumKernel<<<dimGridCumSum, dimBlockCumSum>>>(deviceCbBins, deviceCbHistogramSum);


  //--------- Process 2: YUV Masking
#ifdef USE_STREAMING
  cumSumOne<<<dimGridCumSum, dimBlockCumSum, 0, yuvStream>>>(deviceCbBins, deviceCbOmega, imageWidth * imageHeight, deviceCbHistogramSum);
#else
  cumSumOne<<<dimGridCumSum, dimBlockCumSum>>>(deviceCbBins, deviceCbOmega, imageWidth * imageHeight, deviceCbHistogramSum);
#endif

  //--------- Process 2: YUV Masking
#ifdef USE_STREAMING
  cumSumTwo<<<dimGridCumSum, dimBlockCumSum, 0, yuvStream>>>(deviceCbBins, deviceCbMu, imageWidth * imageHeight, deviceCbHistogramSum);
#else
  cumSumTwo<<<dimGridCumSum, dimBlockCumSum>>>(deviceCbBins, deviceCbMu, imageWidth * imageHeight, deviceCbHistogramSum);
#endif

  //--------- Process 2: YUV Masking
#ifdef USE_STREAMING
  compSigmaBSquared<<<dimGridCumSum, dimBlockCumSum, 0, yuvStream>>>(deviceCbSigmaBSquared, deviceCbOmega, deviceCbMu);
#else
  compSigmaBSquared<<<dimGridCumSum, dimBlockCumSum>>>(deviceCbSigmaBSquared, deviceCbOmega, deviceCbMu);
#endif

  //--------- Process 2: YUV Masking
#ifdef USE_STREAMING
  argmax<<<1, 256, 0, yuvStream>>>(deviceCbThreshold, deviceCbSigmaBSquared);
#else
  argmax<<<1, 256>>>(deviceCbThreshold, deviceCbSigmaBSquared);
#endif

  //--------- Process 2: YUV Masking
#ifdef USE_STREAMING
  maskGeneration<<<dimGridMasking, dimBlockMasking, 0, yuvStream>>>(deviceYUVOutputImageData, deviceCBMaskOutputImageData, deviceCbThreshold, imageWidth, imageHeight, 0, false);
#else
  maskGeneration<<<dimGridMasking, dimBlockMasking>>>(deviceYUVOutputImageData, deviceCBMaskOutputImageData, deviceCbThreshold, imageWidth, imageHeight, 0, false);
#endif
hipDeviceSynchronize();
  hipEventRecord(event3StopEvent, 0);
  hipEventSynchronize(event3StopEvent);
  hipEventElapsedTime(&event3ElapsedTime, event3StartEvent, event3StopEvent);
  printf("Total execution time (ms) %f for YUV Masking \n",event3ElapsedTime);

  //--------- Process 3: Smoothing
 /* dim3 dimGridSmoothing((imageWidth-1)/16 +1, (imageHeight-1)/16+1, 1);
  dim3 dimBlockSmoothing(16, 16, 1);*/

//=======
#ifdef USE_STREAMING
  //smooth_kernel<<<dimGridSmoothing, dimBlockSmoothing, 0, yuvStream>>>(deviceCBMaskOutputImageData, deviceSmoothOutputImageData, deviceMaskData, 1, imageWidth, imageHeight);
  #if SMOOTH_KERNEL_VERSION == 0
  // 2D Shared Memory, Smoothing Kernel
  dim3 dimGridSmoothing((imageWidth * 2 - 1)/16 +1, (imageHeight * 2 -1)/16+1, 1);
  dim3 dimBlockSmoothing(16, 16, 1);
  hipEvent_t event4StartEvent, event4StopEvent;
float event4ElapsedTime;
hipEventCreate(&event4StartEvent);
hipEventCreate(&event4StopEvent);
hipEventRecord(event4StartEvent, 0);
  smooth_kernel<<<dimGridSmoothing, dimBlockSmoothing, 0, yuvStream>>>(deviceCBMaskOutputImageData, deviceSmoothOutputImageData, deviceMaskData, 1, imageWidth, imageHeight);

  #elif SMOOTH_KERNEL_VERSION == 1
  // 1D Shared Memory, Smoothing Kernels (Row + Column)
  smooth_kernel_row<<<dimGridSmoothing, dimBlockSmoothing, 0, yuvStream>>>(deviceCBMaskOutputImageData, deviceSmoothOutputImageData, deviceMaskData, 1, imageWidth, imageHeight);

  smooth_kernel_col<<<dimGridSmoothing, dimBlockSmoothing, 0, yuvStream>>>(deviceSmoothOutputImageData, deviceSmoothOutputImageData, deviceMaskData, 1, imageWidth, imageHeight);

  #elif SMOOTH_KERNEL_VERSION == 2
  // 2D Global Memory, Smoothing Kernel
  smooth_kernel_global<<<dimGridSmoothing, dimBlockSmoothing, 0, yuvStream>>>(deviceCBMaskOutputImageData, deviceSmoothOutputImageData, deviceMaskData, 1, imageWidth, imageHeight); 
  #endif
#else
  //smooth_kernel<<<dimGridSmoothing, dimBlockSmoothing>>>(deviceCBMaskOutputImageData, deviceSmoothOutputImageData, deviceMaskData, 1, imageWidth, imageHeight);
  #if SMOOTH_KERNEL_VERSION == 0
  // 2D Shared Memory, Smoothing Kernel
  dim3 dimGridSmoothing((imageWidth * 2 - 1)/16 +1, (imageHeight * 2 -1)/16+1, 1);
  dim3 dimBlockSmoothing(16, 16, 1);
  smooth_kernel<<<dimGridSmoothing, dimBlockSmoothing>>>(deviceCBMaskOutputImageData, deviceSmoothOutputImageData, deviceMaskData, 1, imageWidth, imageHeight);

  #elif SMOOTH_KERNEL_VERSION == 1
  // 1D Shared Memory, Smoothing Kernels (Row + Column)
  smooth_kernel_row<<<dimGridSmoothing, dimBlockSmoothing>>>(deviceCBMaskOutputImageData, deviceSmoothOutputImageData, deviceMaskData, 1, imageWidth, imageHeight);

  smooth_kernel_col<<<dimGridSmoothing, dimBlockSmoothing>>>(deviceSmoothOutputImageData, deviceSmoothOutputImageData, deviceMaskData, 1, imageWidth, imageHeight);

  #elif SMOOTH_KERNEL_VERSION == 2
  // 2D Global Memory, Smoothing Kernel
  dim3 dimGridSmoothing((imageWidth-1)/16 +1, (imageHeight-1)/16+1, 1);
  dim3 dimBlockSmoothing(16, 16, 1);
  smooth_kernel_global<<<dimGridSmoothing, dimBlockSmoothing>>>(deviceCBMaskOutputImageData, deviceSmoothOutputImageData, deviceMaskData, 1, imageWidth, imageHeight); 
  #endif
#endif
hipDeviceSynchronize();
  hipEventRecord(event4StopEvent, 0);
  hipEventSynchronize(event4StopEvent);
  hipEventElapsedTime(&event4ElapsedTime, event4StartEvent, event4StopEvent);
  printf("Total execution time (ms) %f for smooth kernel\n",event4ElapsedTime);

#if defined(USE_STREAM_EVENTS) && defined(USE_STREAMING)
  hipEventRecord(yuvCompleteEvent, yuvStream);
  hipStreamWaitEvent(resultStream, yuvCompleteEvent, 0);
  hipStreamWaitEvent(resultStream, grayscaleCompleteEvent, 0);
#elif defined(USE_STREAMING)
  hipStreamSynchronize(yuvStream);
  hipStreamSynchronize(grayscaleStream);
#endif

  //--------- Process 5: Ratio & Final Image
  //PROC 5
  hipEvent_t event5StartEvent, event5StopEvent;
  float event5ElapsedTime;
  hipEventCreate(&event5StartEvent);
  hipEventCreate(&event5StopEvent);
  hipEventRecord(event5StartEvent, 0);
#ifdef USE_STREAMING
  map1Proc5<<<dimGridYUVConversion, dimBlockYUVConversion, 0, resultStream>>>(redData, greenData, blueData, deviceErodedShadow, deviceErodedLight, deviceShadowRedArray, deviceShadowGreenArray, deviceShadowBlueArray, deviceLightRedArray, deviceLightGreenArray, deviceLightBlueArray, imageWidth, imageHeight);
#else
  map1Proc5<<<dimGridYUVConversion, dimBlockYUVConversion>>>(redData, greenData, blueData, deviceErodedShadow, deviceErodedLight, deviceShadowRedArray, deviceShadowGreenArray, deviceShadowBlueArray, deviceLightRedArray, deviceLightGreenArray, deviceLightBlueArray, imageWidth, imageHeight);
#endif
  
unsigned int size = imageHeight * imageWidth;

sumProc5Host(deviceErodedShadow, size, deviceErodedShadowSum);
sumProc5Host(deviceErodedLight, size, deviceErodedLightSum);
sumProc5Host(deviceShadowRedArray, size, deviceShadowRedArraySum);
sumProc5Host(deviceShadowGreenArray, size, deviceShadowGreenArraySum);
sumProc5Host(deviceShadowBlueArray, size, deviceShadowBlueArraySum);
sumProc5Host(deviceLightRedArray, size, deviceLightRedArraySum);
sumProc5Host(deviceLightGreenArray, size, deviceLightGreenArraySum);
sumProc5Host(deviceLightBlueArray, size, deviceLightBlueArraySum);

//

//Perform sums

	
#ifdef USE_STREAMING
  smallCalc<<<3,1, 0, resultStream>>>(deviceShadowRedArraySum, deviceShadowGreenArraySum, deviceShadowBlueArraySum, deviceLightRedArraySum, deviceLightGreenArraySum, deviceLightBlueArraySum, deviceErodedLightSum, deviceErodedShadowSum);
#else
  smallCalc<<<3,1>>>(deviceShadowRedArraySum, deviceShadowGreenArraySum, deviceShadowBlueArraySum, deviceLightRedArraySum, deviceLightGreenArraySum, deviceLightBlueArraySum, deviceErodedLightSum, deviceErodedShadowSum);
#endif


#ifdef USE_STREAMING  
  proc5<<<dimGridYUVConversion, dimBlockYUVConversion, 0, resultStream>>>(redData, greenData, blueData, deviceResultImageData, deviceSmoothOutputImageData, deviceShadowRedArraySum, deviceShadowGreenArraySum, deviceShadowBlueArraySum, imageWidth, imageHeight);
#else
  proc5<<<dimGridYUVConversion, dimBlockYUVConversion>>>(redData, greenData, blueData, deviceResultImageData, deviceSmoothOutputImageData, deviceShadowRedArraySum, deviceShadowGreenArraySum, deviceShadowBlueArraySum, imageWidth, imageHeight);
#endif
hipDeviceSynchronize();
hipEventRecord(event5StopEvent, 0);
hipEventSynchronize(event5StopEvent);
hipEventElapsedTime(&event5ElapsedTime, event5StartEvent, event5StopEvent);
printf("Total execution time (ms) %f for result integration \n",event5ElapsedTime);
  

    //hipDeviceSynchronize(); 
  
  //-----------------------------------------------------------------------------------------------------//
  //-------------------------------- Data Transfer (GPU -> Host) ----------------------------------------//
  //-----------------------------------------------------------------------------------------------------//
	
  // YUV Data
  gpuErrchk(hipMemcpy(hostOutputImageDataYUV, deviceYUVOutputImageData,
             imageWidth * imageHeight * imageChannels * sizeof(float),
             hipMemcpyDeviceToHost));

  // Greyscale Data
  gpuErrchk(hipMemcpy(hostOutputImageDataGreyScale, deviceGreyscaleOutputImageData,
             imageWidth * imageHeight * sizeof(float),
             hipMemcpyDeviceToHost));

  // Color Invariant Data
  gpuErrchk(hipMemcpy(hostOutputImageDataColorInvariant, deviceColorInvariantOutputImageData,
             imageWidth * imageHeight * sizeof(float) * imageChannels,
             hipMemcpyDeviceToHost));

  // Mask Data
  gpuErrchk(hipMemcpy(hostOutputImageDataGrayMask, deviceGreyMaskOutputImageData,
             imageWidth * imageHeight * sizeof(float),
             hipMemcpyDeviceToHost));

  gpuErrchk(hipMemcpy(hostOutputImageDataCbMask, deviceCBMaskOutputImageData,
             imageWidth * imageHeight * sizeof(float),
             hipMemcpyDeviceToHost));

  // Smooth Image Data
  gpuErrchk(hipMemcpy(hostOutputImageDataSmooth, deviceSmoothOutputImageData,
	           imageWidth * imageHeight * sizeof(float),
	           hipMemcpyDeviceToHost));

  // Light-mask erosion data

  gpuErrchk(hipMemcpy(hostOutputImageErodedLight, deviceErodedLight,
             imageWidth * imageHeight * sizeof(float),
             hipMemcpyDeviceToHost));

  // Shadow-mask erosion data
  gpuErrchk(hipMemcpy(hostOutputImageErodedShadow, deviceErodedShadow,
             imageWidth * imageHeight * sizeof(float),
             hipMemcpyDeviceToHost));
			 
  // PROC 5
  // Shadow-mask erosion data
  gpuErrchk(hipMemcpy(hostResultImageData, deviceResultImageData,
             imageWidth * imageHeight * imageChannels * sizeof(float),
             hipMemcpyDeviceToHost));

  //-----------------------------------------------------------------------------------------------------//
  //---------------------------------------- Image Export -----------------------------------------------//
  //-----------------------------------------------------------------------------------------------------//

  // wbImage_export expects to be exporting wbImages, so anything we want to export, copy its host buffer into a wbImage
  // YUV Image
  wbImage_t imgOutputYUV = wbImage_new(imageWidth, imageHeight, 3, hostOutputImageDataYUV);
	
  // Grey Scale Image
  wbImage_t imgOutputGreyscale = wbImage_new(imageWidth, imageHeight, 1, hostOutputImageDataGreyScale);
  wbImage_t imgOutputGreyMask = wbImage_new(imageWidth, imageHeight, 1, hostOutputImageDataGrayMask);
  wbImage_t imgOutputCbMask = wbImage_new(imageWidth, imageHeight, 1, hostOutputImageDataCbMask);
	
  // Color Invariant Image
  wbImage_t imgOutputColorInvariant = wbImage_new(imageWidth, imageHeight, imageChannels, hostOutputImageDataColorInvariant);
	
  // Smooth Image
  wbImage_t imgOutputSmooth = wbImage_new(imageWidth, imageHeight, 1, hostOutputImageDataSmooth);
  
  //Result Image
  wbImage_t resultImage = wbImage_new(imageWidth, imageHeight, imageChannels, hostResultImageData);

	
  
  wbImage_t imgOutputLightMaskErosion = wbImage_new(imageWidth, imageHeight, 1, hostOutputImageErodedLight);


  wbImage_t imgOutputShadowMaskErosion = wbImage_new(imageWidth, imageHeight, 1, hostOutputImageErodedShadow);



  // Output Image Path Strings
  std::string yuvOutputPath = baseOutputDir + "Proc1_OutputYUV.ppm";
  std::string greyOutputPath = baseOutputDir + "Proc1_OutputGrey.ppm";
  std::string ciOutputPath = baseOutputDir + "Proc1_OutputCI.ppm";
  
  
  std::string cbMaskOutputPath = baseOutputDir + "Proc2_OutputCbMask.ppm";
  std::string greyMaskOutputPath = baseOutputDir + "Proc2_OutputGreyMask.ppm";

  std::string smoothOutputPath = baseOutputDir + "Proc3_OutputSmooth.ppm";

  std::string erodedLightMaskOutputPath = baseOutputDir + "Proc4_OutputErodedLightMask.ppm";
  std::string erodedShadowMaskOutputPath = baseOutputDir + "Proc4_OutputErodedShadowMask.ppm";
  std::string resultImagePath = baseOutputDir + "Proc5_ResultImage.ppm";

 
  // Export Output Images
  wbExport(yuvOutputPath.c_str(), imgOutputYUV);
  wbExport(greyOutputPath.c_str(), imgOutputGreyscale);
  wbExport(ciOutputPath.c_str(), imgOutputColorInvariant);

  wbExport(cbMaskOutputPath.c_str(), imgOutputCbMask);
  wbExport(greyMaskOutputPath.c_str(), imgOutputGreyMask);
  
  wbExport(smoothOutputPath.c_str(), imgOutputSmooth);

  wbExport(erodedLightMaskOutputPath.c_str(), imgOutputLightMaskErosion);
  wbExport(erodedShadowMaskOutputPath.c_str(), imgOutputShadowMaskErosion);
  wbExport(resultImagePath.c_str(), resultImage);

  // Free Cuda Memory
  hipFree(deviceInputImageData);
  hipFree(deviceYUVOutputImageData);
  hipFree(deviceGreyscaleOutputImageData);
  hipFree(deviceColorInvariantOutputImageData);
  hipFree(deviceGreyMaskOutputImageData);
  hipFree(deviceCBMaskOutputImageData);
  hipFree(deviceSmoothOutputImageData);
  hipFree(deviceMaskData);
  hipFree(deviceErodedLight);
  hipFree(deviceErodedShadow);
  
  // Destroying cuda streams
#ifdef USE_STREAMING
  hipStreamDestroy(colorspaceStream);
  hipStreamDestroy(yuvStream);
  hipStreamDestroy(grayscaleStream);
  hipStreamDestroy(resultStream);
#endif

#if defined(USE_STREAM_EVENTS) && defined(USE_STREAMING)
  hipEventDestroy(colorspaceCompleteEvent);
  hipEventDestroy(yuvCompleteEvent);
  hipEventDestroy(grayscaleCompleteEvent);
#endif

  // Proc 5:
  hipFree(deviceShadowRedArray);
  hipFree(deviceShadowGreenArray);
  hipFree(deviceShadowBlueArray);
  hipFree(deviceLightRedArray);
  hipFree(deviceLightGreenArray);
  hipFree(deviceLightBlueArray);
  
  hipFree(deviceShadowRedArraySum);
  hipFree(deviceShadowGreenArraySum);
  hipFree(deviceShadowBlueArraySum);
  hipFree(deviceLightRedArraySum);
  hipFree(deviceLightGreenArraySum);
  hipFree(deviceLightBlueArraySum);
  hipFree(deviceErodedLightSum);
  hipFree(deviceErodedShadowSum);

	// Delete the Images
  wbImage_delete(inputImage);
  wbImage_delete(imgOutputYUV);
  wbImage_delete(imgOutputGreyscale);
  wbImage_delete(imgOutputColorInvariant);
  wbImage_delete(imgOutputGreyMask);
  wbImage_delete(imgOutputCbMask);
  wbImage_delete(imgOutputSmooth);
  wbImage_delete(imgOutputLightMaskErosion);
  wbImage_delete(imgOutputShadowMaskErosion);
	
  // Print Success
  printf("<('.'<)\n");
  printf("(>'.')>\n");
  printf("\\(-_-)/\n");
  printf("/\\/\\(;;)/\\/\\\n");
  return 0;
}
