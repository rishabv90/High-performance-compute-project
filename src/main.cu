#define _CRT_SECURE_NO_WARNINGS
#include <string>
#include <stdio.h>
#include <wb.h>
#include "RGBtoYUV.cpp"
#include "Greyscale.cpp"
#include "colorInvariant.cpp"
#include "Erosion.cpp"
#include "MaskGeneration.cpp"
#include "Smooth.cpp"
#include "sumProc5.cpp"
#include "map1Proc5.cpp"
#include "proc5.cpp"
#include "map2Proc5.cpp"


#define NUM_BINS 256
#define SMOOTH_KERNEL_VERSION 0	// Define Smooth Version 0 = 2D Shared Memory, 1 = 1D Shared Memory, 2 = 2D Global Memory

#define USE_STREAMING
#define USE_STREAM_EVENTS

//Canonical way to check for errors in CUDA - https://stackoverflow.com/a/14038590
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t errCode, const char *file, int line, bool abort=true) {
  if (errCode != hipSuccess) {
    fprintf(stderr, "GPU Assertion: %s %s %d\n", hipGetErrorString(errCode), file, line);
    if (abort) exit(errCode);
  }
}

static const int erosionStrelWidth = 5, erosionStrelHeight = 5;
static const float hostErosionStrel[erosionStrelHeight][erosionStrelWidth] = {
  {0, 1, 1, 1, 0},
  {1, 1, 1, 1, 1},
  {1, 1, 1, 1, 1},
  {1, 1, 1, 1, 1},
  {0, 1, 1, 1, 0}
};

static const int kernelWidth = 5, kernelHeight = 5;
static const float hostKernelData[kernelWidth][kernelHeight] = {
  {0.04, 0.04, 0.04, 0.04, 0.04},
  {0.04, 0.04, 0.04, 0.04, 0.04},
  {0.04, 0.04, 0.04, 0.04, 0.04},
  {0.04, 0.04, 0.04, 0.04, 0.04},
  {0.04, 0.04, 0.04, 0.04, 0.04}
};

//-------------------------------------------------
//---------------- MAIN FUNCTION:
	// Arguments:
	// ./shadow_removal <- This program call
	// input_image      <- The input image to have shadow removed
	// output_directory <- The directory to produce output images in
	// kernel_file      <- The convolution kernel file

int main(int argc, char** argv) {
  if (argc != 3) {
	fprintf(stderr, "Argument count is: %d and needs to be 3\n", argc);
    fprintf(stderr, "Usage: ./shadow_removal input_image output_directory\n");
    fprintf(stderr, "/(-_-)\\\n");
    return 1;
  }

  //----------------------------------------------------------------------------------------//
  //-------------------------------- Host Variables ----------------------------------------//
  //----------------------------------------------------------------------------------------//
  //--------- Process 0: RGB Image Input
  int imageChannels;
  int imageWidth;
  int imageHeight;
  
  char *inputImageFile = argv[1];
  std::string baseOutputDir(argv[2]);

  if (baseOutputDir.back() != '/') {
    baseOutputDir.append("/");
  }

  wbImage_t inputImage = wbImport(inputImageFile);
  imageWidth = wbImage_getWidth(inputImage);
  imageHeight = wbImage_getHeight(inputImage);
  imageChannels = wbImage_getChannels(inputImage);
  float *hostInputImageData = wbImage_getData(inputImage);

  // -------- Setting up streaming and events for synchronization
#ifdef USE_STREAMING
  hipStream_t colorspaceStream, yuvStream, grayscaleStream, resultStream;
  hipStreamCreate(&colorspaceStream);
  hipStreamCreate(&yuvStream);
  hipStreamCreate(&grayscaleStream);
  hipStreamCreate(&resultStream);
#endif

#if defined(USE_STREAM_EVENTS) && defined(USE_STREAMING)
  hipEvent_t colorspaceCompleteEvent, yuvCompleteEvent, grayscaleCompleteEvent;
  hipEventCreate(&colorspaceCompleteEvent);
  hipEventCreate(&yuvCompleteEvent);
  hipEventCreate(&grayscaleCompleteEvent);
#endif

  //--------- Process 1: YUV Conversion
  float *hostOutputImageDataYUV = (float*) malloc(imageWidth*imageHeight*imageChannels*sizeof(float));

  //--------- Process 1: Color Invariance
  float *hostOutputImageDataColorInvariant = (float*) malloc(imageWidth*imageHeight*imageChannels*sizeof(float));

  //--------- Process 1: Greyscale Conversion
  float *hostOutputImageDataGreyScale = (float*) malloc(imageWidth*imageHeight*sizeof(float));

  //--------- Process 2: YUV Masking
  float *hostOutputImageDataCbMask = (float*) malloc(imageWidth*imageHeight*sizeof(float));

  //--------- Process 2: Greyscale Masking
  float *hostOutputImageDataGrayMask = (float*) malloc(imageWidth*imageHeight*sizeof(float));

  //--------- Process 3: Smoothing
  float *hostOutputImageDataSmooth = (float*) malloc(imageWidth*imageHeight*sizeof(float));

  //--------- Process 4: Light Mask Erosion
  float *hostOutputImageErodedLight = (float*) malloc(imageWidth*imageHeight*sizeof(float));

  //--------- Process 4: Shadow Mask Erosion
  float *hostOutputImageErodedShadow = (float*) malloc(imageWidth*imageHeight*sizeof(float));

  //--------- Process 5: Ratio & Final Image
  float *hostResultImageData = (float*) malloc(imageWidth*imageHeight*imageChannels*sizeof(float));

  //------------------------------------------------------------------------------------------//
  //-------------------------------- Device Variables ----------------------------------------//
  //------------------------------------------------------------------------------------------//
  //--------- Process 0: RGB Image Input
  float *deviceInputImageData;
  gpuErrchk(hipMalloc((void **) &deviceInputImageData, imageWidth * imageHeight * imageChannels * sizeof(float)));

  //--------- Process 1: YUV Conversion
  float *deviceYUVOutputImageData;
  gpuErrchk(hipMalloc((void **) &deviceYUVOutputImageData, imageWidth * imageHeight * imageChannels * sizeof(float)));

  //--------- Process 1: Color Invariance
  float *deviceColorInvariantOutputImageData;
  gpuErrchk(hipMalloc((void **)&deviceColorInvariantOutputImageData, imageWidth * imageHeight * sizeof(float) * imageChannels));

  //--------- Process 1: Greyscale Conversion
  float *deviceGreyscaleOutputImageData;
  float *redData;
  float *greenData;
  float *blueData;
  gpuErrchk(hipMalloc((void **)&deviceGreyscaleOutputImageData, imageWidth * imageHeight * sizeof(float)));
  
  //For coalesced accesses after first process
  gpuErrchk(hipMalloc((void **)&redData, imageWidth * imageHeight * sizeof(float)));
  gpuErrchk(hipMalloc((void **)&greenData, imageWidth * imageHeight * sizeof(float)));
  gpuErrchk(hipMalloc((void **)&blueData, imageWidth * imageHeight * sizeof(float)));

  //--------- Process 2: YUV Masking
  float *deviceCBMaskOutputImageData;
  unsigned int *deviceCbBins;
  unsigned int *histogramSum;
  float *deviceGrayOmega;
  float *deviceCbOmega;
  float *deviceGrayMu;
  float *deviceCbMu;
  float *deviceGraySigmaBSquared;
  float *deviceCbSigmaBSquared;
  float *deviceGrayThreshold;
  float *deviceCbThreshold;
  
  gpuErrchk(hipMalloc((void **)&deviceCBMaskOutputImageData, imageWidth * imageHeight * sizeof(float)));
  gpuErrchk(hipMalloc((void **)&deviceCbBins, NUM_BINS * sizeof(int)));
  gpuErrchk(hipMalloc((void **)&histogramSum, sizeof(int)));
  gpuErrchk(hipMalloc((void **)&deviceGrayOmega, NUM_BINS * sizeof(float)));
  gpuErrchk(hipMalloc((void **)&deviceCbOmega, NUM_BINS * sizeof(float)));
  gpuErrchk(hipMalloc((void **)&deviceGrayMu, NUM_BINS * sizeof(float)));
  gpuErrchk(hipMalloc((void **)&deviceCbMu, NUM_BINS * sizeof(float)));
  gpuErrchk(hipMalloc((void **)&deviceGraySigmaBSquared, NUM_BINS * sizeof(float)));
  gpuErrchk(hipMalloc((void **)&deviceCbSigmaBSquared, NUM_BINS * sizeof(float)));
  gpuErrchk(hipMalloc((void **)&deviceGrayThreshold, sizeof(float)));
  gpuErrchk(hipMalloc((void **)&deviceCbThreshold, sizeof(float)));

  //--------- Process 2: Greyscale Masking
  float *deviceGreyMaskOutputImageData;
  unsigned int *deviceGreyBins;

  gpuErrchk(hipMalloc((void **)&deviceGreyMaskOutputImageData, imageWidth * imageHeight * sizeof(float))); 
  gpuErrchk(hipMalloc((void **)&deviceGreyBins, NUM_BINS * sizeof(int)));

  //--------- Process 3: Smoothing
  float *deviceSmoothOutputImageData;
  float *deviceMaskData;

  gpuErrchk(hipMalloc((void **)&deviceSmoothOutputImageData, imageWidth * imageHeight * sizeof(float)));
  gpuErrchk(hipMalloc((void **)&deviceMaskData, kernelWidth * kernelHeight * sizeof(float)));

  //--------- Process 4: Light Mask Erosion
  float *deviceErodedLight;
  float *deviceStrel;

  gpuErrchk(hipMalloc((void **)&deviceErodedLight, imageHeight * imageWidth * sizeof(float)));
  gpuErrchk(hipMalloc((void **)&deviceStrel, erosionStrelHeight * erosionStrelWidth * sizeof(float)));

  //--------- Process 4: Shadow Mask Erosion
  float *deviceErodedShadow;

  gpuErrchk(hipMalloc((void **)&deviceErodedShadow, imageHeight * imageWidth * sizeof(float)));

  //--------- Process 5: Final Image
  float *deviceShadowRedArray;
  float *deviceShadowGreenArray;
  float *deviceShadowBlueArray;
  float *deviceLightRedArray;
  float *deviceLightGreenArray;
  float *deviceLightBlueArray;
  float *deviceResultImageData;
  
  gpuErrchk(hipMalloc((void **)&deviceShadowRedArray, imageHeight * imageWidth * sizeof(float)));
  gpuErrchk(hipMalloc((void **)&deviceShadowGreenArray, imageHeight * imageWidth * sizeof(float)));
  gpuErrchk(hipMalloc((void **)&deviceShadowBlueArray, imageHeight * imageWidth * sizeof(float)));
  gpuErrchk(hipMalloc((void **)&deviceLightRedArray, imageHeight * imageWidth * sizeof(float)));
  gpuErrchk(hipMalloc((void **)&deviceLightGreenArray, imageHeight * imageWidth * sizeof(float)));
  gpuErrchk(hipMalloc((void **)&deviceLightBlueArray, imageHeight * imageWidth * sizeof(float)));
  gpuErrchk(hipMalloc((void **)&deviceResultImageData, imageHeight * imageWidth * imageChannels * sizeof(float)));


  //-----------------------------------------------------------------------------------------------------//
  //-------------------------------- Data Transfer (Host -> GPU) ----------------------------------------//
  //-----------------------------------------------------------------------------------------------------//
  gpuErrchk(hipMemcpy(deviceInputImageData, hostInputImageData,
             imageWidth * imageHeight * imageChannels * sizeof(float),
             hipMemcpyHostToDevice));	 
			 
  #ifndef USE_STREAMING
  gpuErrchk(hipMemcpy(deviceMaskData, hostKernelData,
             kernelWidth * kernelHeight * sizeof(float), 
			       hipMemcpyHostToDevice));

  gpuErrchk(hipMemcpy(deviceStrel, hostErosionStrel,
             erosionStrelWidth * erosionStrelHeight * sizeof(float),
             hipMemcpyHostToDevice));
  #else
  gpuErrchk(hipMemcpyAsync(deviceMaskData, hostKernelData,
             kernelWidth * kernelHeight * sizeof(float),
             hipMemcpyHostToDevice, yuvStream));

  gpuErrchk(hipMemcpyAsync(deviceStrel, hostErosionStrel,
             erosionStrelWidth * erosionStrelHeight * sizeof(float),
             hipMemcpyHostToDevice, grayscaleStream));
  #endif


  //-------------------------------------------------------------------------------------------------//
  //-------------------------------- STOP! IT'S KERNEL TIME! ----------------------------------------//
  //-------------------------------------------------------------------------------------------------//

  // Shared grid or block dimension tuples
  dim3 dimGridHisto(2, 1, 1);
  dim3 dimBlockHisto(1024, 1, 1);

  dim3 dimGridCumSum(1, 1, 1);
  dim3 dimBlockCumSum(NUM_BINS, 1, 1);

  dim3 dimGridMasking((imageWidth - 1) / 16 + 1, (imageHeight - 1)/16 + 1, 1);
  dim3 dimBlockMasking(16, 16, 1);

  dim3 dimGridErosion((imageWidth - 1) / 16 + 1, (imageHeight - 1)/16 + 1, 1);
  dim3 dimBlockErosion(16, 16, 1);

  // Kernel Launches:
  // Color Invariant -> Grayscale processing, and YUV
  //--------- Process 1: Color Invariance
  dim3 dimGridColorInvariance((imageWidth - 1) / 16 + 1, (imageHeight - 1)/16 + 1, 1);
  dim3 dimBlockColorInvariance(16, 16, 1);

#ifdef USE_STREAMING
  YUVandCItoGray<<<dimGridColorInvariance, dimBlockColorInvariance, 0, colorspaceStream>>>(deviceGreyscaleOutputImageData, deviceYUVOutputImageData, deviceInputImageData, redData, greenData, blueData, imageWidth, imageHeight); //implemented
#else
  YUVandCItoGray<<<dimGridColorInvariance, dimBlockColorInvariance>>>(deviceGreyscaleOutputImageData, deviceYUVOutputImageData, deviceInputImageData, redData, greenData, blueData, imageWidth, imageHeight);
#endif
  // Wait until this stream is done as all other streams depend on the colorspace transform
#if defined(USE_STREAM_EVENTS) && defined(USE_STREAMING)
  hipEventRecord(colorspaceCompleteEvent, colorspaceStream);
  hipStreamWaitEvent(grayscaleStream, colorspaceCompleteEvent, 0);
#elif defined(USE_STREAMING)
  hipStreamSynchronize(colorspaceStream);
#endif
  /*
  //--------- Process 1: Greyscale Conversion, done previously

  
  //--------- Process 2: Greyscale Masking
#ifdef USE_STREAMING
  histogramKernel<<<dimGridHisto, dimBlockHisto, 0, grayscaleStream>>>(deviceGreyscaleOutputImageData, deviceGreyBins, imageWidth * imageHeight); //implemented
#else
  histogramKernel<<<dimGridHisto, dimBlockHisto>>>(deviceGreyscaleOutputImageData, deviceGreyBins, imageWidth * imageHeight); 
#endif
  
  //--------- Process 2: Greyscale Masking
#ifdef USE_STREAMING
  cumSumOne<<<dimGridCumSum, dimBlockCumSum, 0, grayscaleStream>>>(deviceGreyBins, deviceGrayOmega, imageWidth * imageHeight); //pending
#else
  cumSumOne<<<dimGridCumSum, dimBlockCumSum>>>(deviceGreyBins, deviceGrayOmega, imageWidth * imageHeight);
#endif
  
  //--------- Process 2: Greyscale Masking
#ifdef USE_STREAMING
  cumSumTwo<<<dimGridCumSum, dimBlockCumSum, 0, grayscaleStream>>>(deviceGreyBins, deviceGrayMu, imageWidth * imageHeight); //pending
#else
  cumSumTwo<<<dimGridCumSum, dimBlockCumSum>>>(deviceGreyBins, deviceGrayMu, imageWidth * imageHeight);
#endif
  
  //--------- Process 2: Greyscale Masking
#ifdef USE_STREAMING
  compSigmaBSquared<<<dimGridCumSum, dimBlockCumSum, 0, grayscaleStream>>>(deviceGraySigmaBSquared, deviceGrayOmega, deviceGrayMu);
#else
  compSigmaBSquared<<<dimGridCumSum, dimBlockCumSum>>>(deviceGraySigmaBSquared, deviceGrayOmega, deviceGrayMu);
#endif
  //--------- Process 2: Greyscale Masking
#ifdef USE_STREAMING
  argmax<<<1, 256, 0, grayscaleStream>>>(deviceGrayThreshold, deviceGraySigmaBSquared); //pending
#else
  argmax<<<1, 256>>>(deviceGrayThreshold, deviceGraySigmaBSquared);
#endif
  
  //--------- Process 2: Greyscale Masking
#ifdef USE_STREAMING
  maskGeneration<<<dimGridMasking, dimBlockMasking, 0, grayscaleStream>>>(deviceGreyscaleOutputImageData, deviceGreyMaskOutputImageData, deviceGrayThreshold, imageWidth, imageHeight, 1);
#else
  maskGeneration<<<dimGridMasking, dimBlockMasking>>>(deviceGreyscaleOutputImageData, deviceGreyMaskOutputImageData, deviceGrayThreshold, imageWidth, imageHeight, 1);
#endif

  //--------- Process 4: Light Mask Erosion
#ifdef USE_STREAMING
  maskErosion<<<dimGridErosion, dimBlockErosion, 0, grayscaleStream>>>(deviceErodedLight, deviceGreyMaskOutputImageData, deviceStrel, imageWidth, imageHeight, true);
#else
  maskErosion<<<dimGridErosion, dimBlockErosion>>>(deviceErodedLight, deviceGreyMaskOutputImageData, deviceStrelLight, imageWidth, imageHeight, true);
#endif

  //--------- Process 4: Shadow Mask Erosion
#ifdef USE_STREAMING  
  maskErosion<<<dimGridErosion, dimBlockErosion, 0, grayscaleStream>>>(deviceErodedShadow, deviceGreyMaskOutputImageData, deviceStrel, imageWidth, imageHeight, false);
#else
  maskErosion<<<dimGridErosion, dimBlockErosion>>>(deviceErodedShadow, deviceGreyMaskOutputImageData, deviceStrelShadow, imageWidth, imageHeight, false);
#endif*/

//START HERE

#if defined(USE_STREAM_EVENTS) && defined(USE_STREAMING)
  hipEventRecord(grayscaleCompleteEvent, grayscaleStream);
  hipStreamWaitEvent(yuvStream, colorspaceCompleteEvent, 0);
#endif
  // YUV processing
  //--------- Process 1: YUV Conversion, done previously
  dim3 dimGridYUVConversion((imageWidth - 1) / 16 + 1, (imageHeight - 1)/16 + 1, 1);
  dim3 dimBlockYUVConversion(16, 16, 1);


  //--------- Process 2: YUV Masking
#ifdef USE_STREAMING
  histogramKernel<<<dimGridHisto, dimBlockHisto, 0, yuvStream>>>(deviceYUVOutputImageData, deviceCbBins, imageWidth * imageHeight); //implemented
#else
  histogramKernel<<<dimGridHisto, dimBlockHisto>>>(deviceYUVOutputImageData, deviceCbBins, imageWidth * imageHeight);
#endif

histogramSumKernel<<<dimGridCumSum, dimBlockCumSum>>>(deviceCbBins, histogramSum);


  //--------- Process 2: YUV Masking
#ifdef USE_STREAMING
  cumSumOne<<<dimGridCumSum, dimBlockCumSum, 0, yuvStream>>>(deviceCbBins, deviceCbOmega, imageWidth * imageHeight, histogramSum);
#else
  cumSumOne<<<dimGridCumSum, dimBlockCumSum>>>(deviceCbBins, deviceCbOmega, imageWidth * imageHeight, histogramSum);
#endif

  //--------- Process 2: YUV Masking
#ifdef USE_STREAMING
  cumSumTwo<<<dimGridCumSum, dimBlockCumSum, 0, yuvStream>>>(deviceCbBins, deviceCbMu, imageWidth * imageHeight, histogramSum);
#else
  cumSumTwo<<<dimGridCumSum, dimBlockCumSum>>>(deviceCbBins, deviceCbMu, imageWidth * imageHeight, histogramSum);
#endif

  //--------- Process 2: YUV Masking
#ifdef USE_STREAMING
  compSigmaBSquared<<<dimGridCumSum, dimBlockCumSum, 0, yuvStream>>>(deviceCbSigmaBSquared, deviceCbOmega, deviceCbMu);
#else
  compSigmaBSquared<<<dimGridCumSum, dimBlockCumSum>>>(deviceCbSigmaBSquared, deviceCbOmega, deviceCbMu);
#endif

  //--------- Process 2: YUV Masking
#ifdef USE_STREAMING
  argmax<<<1, 256, 0, yuvStream>>>(deviceCbThreshold, deviceCbSigmaBSquared);
#else
  argmax<<<1, 256>>>(deviceCbThreshold, deviceCbSigmaBSquared);
#endif

  //--------- Process 2: YUV Masking
#ifdef USE_STREAMING
  maskGeneration<<<dimGridMasking, dimBlockMasking, 0, yuvStream>>>(deviceYUVOutputImageData, deviceCBMaskOutputImageData, deviceCbThreshold, imageWidth, imageHeight, 0);
#else
  maskGeneration<<<dimGridMasking, dimBlockMasking>>>(deviceYUVOutputImageData, deviceCBMaskOutputImageData, deviceCbThreshold, imageWidth, imageHeight, 0);
#endif

  //--------- Process 3: Smoothing
 /* dim3 dimGridSmoothing((imageWidth-1)/16 +1, (imageHeight-1)/16+1, 1);
  dim3 dimBlockSmoothing(16, 16, 1);*/

//=======
#ifdef USE_STREAMING
  //smooth_kernel<<<dimGridSmoothing, dimBlockSmoothing, 0, yuvStream>>>(deviceCBMaskOutputImageData, deviceSmoothOutputImageData, deviceMaskData, 1, imageWidth, imageHeight);
  #if SMOOTH_KERNEL_VERSION == 0
  // 2D Shared Memory, Smoothing Kernel
  dim3 dimGridSmoothing((imageWidth * 2 - 1)/16 +1, (imageHeight * 2 -1)/16+1, 1);
  dim3 dimBlockSmoothing(16, 16, 1);
  smooth_kernel<<<dimGridSmoothing, dimBlockSmoothing, 0, yuvStream>>>(deviceCBMaskOutputImageData, deviceSmoothOutputImageData, deviceMaskData, 1, imageWidth, imageHeight);

  #elif SMOOTH_KERNEL_VERSION == 1
  // 1D Shared Memory, Smoothing Kernels (Row + Column)
  smooth_kernel_row<<<dimGridSmoothing, dimBlockSmoothing, 0, yuvStream>>>(deviceCBMaskOutputImageData, deviceSmoothOutputImageData, deviceMaskData, 1, imageWidth, imageHeight);

  smooth_kernel_col<<<dimGridSmoothing, dimBlockSmoothing, 0, yuvStream>>>(deviceSmoothOutputImageData, deviceSmoothOutputImageData, deviceMaskData, 1, imageWidth, imageHeight);

  #elif SMOOTH_KERNEL_VERSION == 2
  // 2D Global Memory, Smoothing Kernel
  smooth_kernel_global<<<dimGridSmoothing, dimBlockSmoothing, 0, yuvStream>>>(deviceCBMaskOutputImageData, deviceSmoothOutputImageData, deviceMaskData, 1, imageWidth, imageHeight); 
  #endif
#else
  //smooth_kernel<<<dimGridSmoothing, dimBlockSmoothing>>>(deviceCBMaskOutputImageData, deviceSmoothOutputImageData, deviceMaskData, 1, imageWidth, imageHeight);
  #if SMOOTH_KERNEL_VERSION == 0
  // 2D Shared Memory, Smoothing Kernel
  dim3 dimGridSmoothing((imageWidth * 2 - 1)/16 +1, (imageHeight * 2 -1)/16+1, 1);
  dim3 dimBlockSmoothing(16, 16, 1);
  smooth_kernel<<<dimGridSmoothing, dimBlockSmoothing>>>(deviceCBMaskOutputImageData, deviceSmoothOutputImageData, deviceMaskData, 1, imageWidth, imageHeight);

  #elif SMOOTH_KERNEL_VERSION == 1
  // 1D Shared Memory, Smoothing Kernels (Row + Column)
  smooth_kernel_row<<<dimGridSmoothing, dimBlockSmoothing>>>(deviceCBMaskOutputImageData, deviceSmoothOutputImageData, deviceMaskData, 1, imageWidth, imageHeight);

  smooth_kernel_col<<<dimGridSmoothing, dimBlockSmoothing>>>(deviceSmoothOutputImageData, deviceSmoothOutputImageData, deviceMaskData, 1, imageWidth, imageHeight);

  #elif SMOOTH_KERNEL_VERSION == 2
  // 2D Global Memory, Smoothing Kernel
  dim3 dimGridSmoothing((imageWidth-1)/16 +1, (imageHeight-1)/16+1, 1);
  dim3 dimBlockSmoothing(16, 16, 1);
  smooth_kernel_global<<<dimGridSmoothing, dimBlockSmoothing>>>(deviceCBMaskOutputImageData, deviceSmoothOutputImageData, deviceMaskData, 1, imageWidth, imageHeight); 
  #endif
#endif


#if defined(USE_STREAM_EVENTS) && defined(USE_STREAMING)
  hipEventRecord(yuvCompleteEvent, yuvStream);
  hipStreamWaitEvent(resultStream, yuvCompleteEvent, 0);
  hipStreamWaitEvent(resultStream, grayscaleCompleteEvent, 0);
#elif defined(USE_STREAMING)
  hipStreamSynchronize(yuvStream);
  hipStreamSynchronize(grayscaleStream);
#endif

  //--------- Process 5: Ratio & Final Image
  //PROC 5
#ifdef USE_STREAMING
  map1Proc5<<<dimGridYUVConversion, dimBlockYUVConversion, 0, resultStream>>>(redData, greenData, blueData, deviceErodedShadow, deviceErodedLight, deviceShadowRedArray, deviceShadowGreenArray, deviceShadowBlueArray, deviceLightRedArray, deviceLightGreenArray, deviceLightBlueArray, imageWidth, imageHeight);
#else
  map1Proc5<<<dimGridYUVConversion, dimBlockYUVConversion>>>(redData, greenData, blueData, deviceErodedShadow, deviceErodedLight, deviceShadowRedArray, deviceShadowGreenArray, deviceShadowBlueArray, deviceLightRedArray, deviceLightGreenArray, deviceLightBlueArray, imageWidth, imageHeight);
#endif
  
    //sum
    int size = imageWidth * imageHeight;
    int maxThreadsPerBlock = 1024;
    int maxThreadsPerBlock2 = maxThreadsPerBlock*2;
    int blocks = ((size - 1) / maxThreadsPerBlock2) + 1;

#ifdef USE_STREAMING
    sumProc5<<<blocks, maxThreadsPerBlock, maxThreadsPerBlock*8*sizeof(float), resultStream>>>(deviceErodedShadow, deviceErodedLight, deviceShadowRedArray, deviceShadowGreenArray, deviceShadowBlueArray, deviceLightRedArray, deviceLightGreenArray, deviceLightBlueArray, deviceErodedShadow, deviceErodedLight, deviceShadowRedArray, deviceShadowGreenArray, deviceShadowBlueArray, deviceLightRedArray, deviceLightGreenArray, deviceLightBlueArray, size);//in fist par deviceGreyscaleOutputImageData - test
#else
    sumProc5<<<blocks, maxThreadsPerBlock, maxThreadsPerBlock*8*sizeof(float)>>>(deviceErodedShadow, deviceErodedLight, deviceShadowRedArray, deviceShadowGreenArray, deviceShadowBlueArray, deviceLightRedArray, deviceLightGreenArray, deviceLightBlueArray, deviceErodedShadow, deviceErodedLight, deviceShadowRedArray, deviceShadowGreenArray, deviceShadowBlueArray, deviceLightRedArray, deviceLightGreenArray, deviceLightBlueArray, size);//in fist par deviceGreyscaleOutputImageData - test
#endif
    size = blocks;
    blocks = ((blocks-1) / maxThreadsPerBlock2) + 1;
    if(size > maxThreadsPerBlock2){
#ifdef USE_STREAMING
    sumProc5<<<blocks, maxThreadsPerBlock, maxThreadsPerBlock*8*sizeof(float), resultStream>>>(deviceErodedShadow, deviceErodedLight, deviceShadowRedArray, deviceShadowGreenArray, deviceShadowBlueArray, deviceLightRedArray, deviceLightGreenArray, deviceLightBlueArray, deviceErodedShadow, deviceErodedLight, deviceShadowRedArray, deviceShadowGreenArray, deviceShadowBlueArray, deviceLightRedArray, deviceLightGreenArray, deviceLightBlueArray, size);//in fist par deviceGreyscaleOutputImageData - test
#else
    sumProc5<<<blocks, maxThreadsPerBlock, maxThreadsPerBlock*8*sizeof(float)>>>(deviceErodedShadow, deviceErodedLight, deviceShadowRedArray, deviceShadowGreenArray, deviceShadowBlueArray, deviceLightRedArray, deviceLightGreenArray, deviceLightBlueArray, deviceErodedShadow, deviceErodedLight, deviceShadowRedArray, deviceShadowGreenArray, deviceShadowBlueArray, deviceLightRedArray, deviceLightGreenArray, deviceLightBlueArray, size);//in fist par deviceGreyscaleOutputImageData - test
#endif
      size = blocks;
      blocks = ((blocks-1) / maxThreadsPerBlock2) + 1;
#ifdef USE_STREAMING
    sumProc5<<<blocks, maxThreadsPerBlock, maxThreadsPerBlock*8*sizeof(float), resultStream>>>(deviceErodedShadow, deviceErodedLight, deviceShadowRedArray, deviceShadowGreenArray, deviceShadowBlueArray, deviceLightRedArray, deviceLightGreenArray, deviceLightBlueArray, deviceErodedShadow, deviceErodedLight, deviceShadowRedArray, deviceShadowGreenArray, deviceShadowBlueArray, deviceLightRedArray, deviceLightGreenArray, deviceLightBlueArray, size);//in fist par deviceGreyscaleOutputImageData - test
#else
    sumProc5<<<blocks, maxThreadsPerBlock, maxThreadsPerBlock*8*sizeof(float)>>>(deviceErodedShadow, deviceErodedLight, deviceShadowRedArray, deviceShadowGreenArray, deviceShadowBlueArray, deviceLightRedArray, deviceLightGreenArray, deviceLightBlueArray, deviceErodedShadow, deviceErodedLight, deviceShadowRedArray, deviceShadowGreenArray, deviceShadowBlueArray, deviceLightRedArray, deviceLightGreenArray, deviceLightBlueArray, size);//in fist par deviceGreyscaleOutputImageData - test
#endif
    }
    else{

#ifdef USE_STREAMING
    sumProc5<<<blocks, maxThreadsPerBlock, maxThreadsPerBlock*8*sizeof(float), resultStream>>>(deviceErodedShadow, deviceErodedLight, deviceShadowRedArray, deviceShadowGreenArray, deviceShadowBlueArray, deviceLightRedArray, deviceLightGreenArray, deviceLightBlueArray, deviceErodedShadow, deviceErodedLight, deviceShadowRedArray, deviceShadowGreenArray, deviceShadowBlueArray, deviceLightRedArray, deviceLightGreenArray, deviceLightBlueArray, size);//in fist par deviceGreyscaleOutputImageData - test
#else
    sumProc5<<<blocks, maxThreadsPerBlock, maxThreadsPerBlock*8*sizeof(float)>>>(deviceErodedShadow, deviceErodedLight, deviceShadowRedArray, deviceShadowGreenArray, deviceShadowBlueArray, deviceLightRedArray, deviceLightGreenArray, deviceLightBlueArray, deviceErodedShadow, deviceErodedLight, deviceShadowRedArray, deviceShadowGreenArray, deviceShadowBlueArray, deviceLightRedArray, deviceLightGreenArray, deviceLightBlueArray, size);//in fist par deviceGreyscaleOutputImageData - test
#endif

    }
	
#ifdef USE_STREAMING
  smallCalc<<<3,1, 0, resultStream>>>(deviceShadowRedArray, deviceShadowGreenArray, deviceShadowBlueArray, deviceLightRedArray, deviceLightGreenArray, deviceLightBlueArray, deviceErodedLight, deviceErodedShadow);
#else
  smallCalc<<<3,1>>>(deviceShadowRedArray, deviceShadowGreenArray, deviceShadowBlueArray, deviceLightRedArray, deviceLightGreenArray, deviceLightBlueArray, deviceErodedLight, deviceErodedShadow);
#endif


#ifdef USE_STREAMING  
  proc5<<<dimGridYUVConversion, dimBlockYUVConversion, 0, resultStream>>>(redData, greenData, blueData, deviceResultImageData, deviceSmoothOutputImageData, deviceShadowRedArray, deviceShadowGreenArray, deviceShadowBlueArray, imageWidth, imageHeight);
#else
  proc5<<<dimGridYUVConversion, dimBlockYUVConversion>>>(redData, greenData, blueData, deviceResultImageData, deviceSmoothOutputImageData, deviceShadowRedArray, deviceShadowGreenArray, deviceShadowBlueArray, imageWidth, imageHeight);
#endif
	
  

    //hipDeviceSynchronize(); 
  
  //-----------------------------------------------------------------------------------------------------//
  //-------------------------------- Data Transfer (GPU -> Host) ----------------------------------------//
  //-----------------------------------------------------------------------------------------------------//
	
  // YUV Data
  gpuErrchk(hipMemcpy(hostOutputImageDataYUV, deviceYUVOutputImageData,
             imageWidth * imageHeight * imageChannels * sizeof(float),
             hipMemcpyDeviceToHost));

  // Greyscale Data
  gpuErrchk(hipMemcpy(hostOutputImageDataGreyScale, deviceGreyscaleOutputImageData,
             imageWidth * imageHeight * sizeof(float),
             hipMemcpyDeviceToHost));

  // Color Invariant Data
  gpuErrchk(hipMemcpy(hostOutputImageDataColorInvariant, deviceColorInvariantOutputImageData,
             imageWidth * imageHeight * sizeof(float) * imageChannels,
             hipMemcpyDeviceToHost));

  // Mask Data
  gpuErrchk(hipMemcpy(hostOutputImageDataGrayMask, deviceGreyMaskOutputImageData,
             imageWidth * imageHeight * sizeof(float),
             hipMemcpyDeviceToHost));

  gpuErrchk(hipMemcpy(hostOutputImageDataCbMask, deviceCBMaskOutputImageData,
             imageWidth * imageHeight * sizeof(float),
             hipMemcpyDeviceToHost));

  // Smooth Image Data
  gpuErrchk(hipMemcpy(hostOutputImageDataSmooth, deviceSmoothOutputImageData,
	           imageWidth * imageHeight * sizeof(float),
	           hipMemcpyDeviceToHost));

  // Light-mask erosion data
  gpuErrchk(hipMemcpy(hostOutputImageErodedLight, deviceErodedLight,
             imageWidth * imageHeight * sizeof(float),
             hipMemcpyDeviceToHost));

  // Shadow-mask erosion data
  gpuErrchk(hipMemcpy(hostOutputImageErodedShadow, deviceErodedShadow,
             imageWidth * imageHeight * sizeof(float),
             hipMemcpyDeviceToHost));
			 
  // PROC 5
  // Shadow-mask erosion data
  gpuErrchk(hipMemcpy(hostResultImageData, deviceResultImageData,
             imageWidth * imageHeight * imageChannels * sizeof(float),
             hipMemcpyDeviceToHost));

  //-----------------------------------------------------------------------------------------------------//
  //---------------------------------------- Image Export -----------------------------------------------//
  //-----------------------------------------------------------------------------------------------------//

  // wbImage_export expects to be exporting wbImages, so anything we want to export, copy its host buffer into a wbImage
  // YUV Image
  wbImage_t imgOutputYUV = wbImage_new(imageWidth, imageHeight, 3, hostOutputImageDataYUV);
	
  // Grey Scale Image
  wbImage_t imgOutputGreyscale = wbImage_new(imageWidth, imageHeight, 1, hostOutputImageDataGreyScale);
  wbImage_t imgOutputGreyMask = wbImage_new(imageWidth, imageHeight, 1, hostOutputImageDataGrayMask);
  wbImage_t imgOutputCbMask = wbImage_new(imageWidth, imageHeight, 1, hostOutputImageDataCbMask);
	
  // Color Invariant Image
  wbImage_t imgOutputColorInvariant = wbImage_new(imageWidth, imageHeight, imageChannels, hostOutputImageDataColorInvariant);
	
  // Smooth Image
  wbImage_t imgOutputSmooth = wbImage_new(imageWidth, imageHeight, 1, hostOutputImageDataSmooth);
  
  //Result Image
  wbImage_t resultImage = wbImage_new(imageWidth, imageHeight, imageChannels, hostResultImageData);

	
  // Erosion images
  wbImage_t imgOutputLightMaskErosion = wbImage_new(imageWidth, imageHeight, 1, hostOutputImageErodedLight);
  wbImage_t imgOutputShadowMaskErosion = wbImage_new(imageWidth, imageHeight, 1, hostOutputImageErodedShadow);

  // Output Image Path Strings
  std::string yuvOutputPath = baseOutputDir + "Proc1_OutputYUV.ppm";
  std::string greyOutputPath = baseOutputDir + "Proc1_OutputGrey.ppm";
  std::string ciOutputPath = baseOutputDir + "Proc1_OutputCI.ppm";
  
  
  std::string cbMaskOutputPath = baseOutputDir + "Proc2_OutputCbMask.ppm";
  std::string greyMaskOutputPath = baseOutputDir + "Proc2_OutputGreyMask.ppm";

  std::string smoothOutputPath = baseOutputDir + "Proc3_OutputSmooth.ppm";

  std::string erodedLightMaskOutputPath = baseOutputDir + "Proc4_OutputErodedLightMask.ppm";
  std::string erodedShadowMaskOutputPath = baseOutputDir + "Proc4_OutputErodedShadowMask.ppm";
  std::string resultImagePath = baseOutputDir + "Proc5_ResultImage.ppm";

 
  // Export Output Images
  wbExport(yuvOutputPath.c_str(), imgOutputYUV);
  wbExport(greyOutputPath.c_str(), imgOutputGreyscale);
  wbExport(ciOutputPath.c_str(), imgOutputColorInvariant);

  wbExport(cbMaskOutputPath.c_str(), imgOutputCbMask);
  wbExport(greyMaskOutputPath.c_str(), imgOutputGreyMask);
  
  wbExport(smoothOutputPath.c_str(), imgOutputSmooth);

  wbExport(erodedLightMaskOutputPath.c_str(), imgOutputLightMaskErosion);
  wbExport(erodedShadowMaskOutputPath.c_str(), imgOutputShadowMaskErosion);
  wbExport(resultImagePath.c_str(), resultImage);

  // Free Cuda Memory
  hipFree(deviceInputImageData);
  hipFree(deviceYUVOutputImageData);
  hipFree(deviceGreyscaleOutputImageData);
  hipFree(deviceColorInvariantOutputImageData);
  hipFree(deviceGreyMaskOutputImageData);
  hipFree(deviceCBMaskOutputImageData);
  hipFree(deviceSmoothOutputImageData);
  hipFree(deviceMaskData);
  hipFree(deviceErodedLight);
  hipFree(deviceErodedShadow);
  
  // Destroying cuda streams
#ifdef USE_STREAMING
  hipStreamDestroy(colorspaceStream);
  hipStreamDestroy(yuvStream);
  hipStreamDestroy(grayscaleStream);
  hipStreamDestroy(resultStream);
#endif

#if defined(USE_STREAM_EVENTS) && defined(USE_STREAMING)
  hipEventDestroy(colorspaceCompleteEvent);
  hipEventDestroy(yuvCompleteEvent);
  hipEventDestroy(grayscaleCompleteEvent);
#endif

  // Proc 5:
  hipFree(deviceShadowRedArray);
  hipFree(deviceShadowGreenArray);
  hipFree(deviceShadowBlueArray);
  hipFree(deviceLightRedArray);
  hipFree(deviceLightGreenArray);
  hipFree(deviceLightBlueArray);

	// Delete the Images
  wbImage_delete(inputImage);
  wbImage_delete(imgOutputYUV);
  wbImage_delete(imgOutputGreyscale);
  wbImage_delete(imgOutputColorInvariant);
  wbImage_delete(imgOutputGreyMask);
  wbImage_delete(imgOutputCbMask);
  wbImage_delete(imgOutputSmooth);
  wbImage_delete(imgOutputLightMaskErosion);
  wbImage_delete(imgOutputShadowMaskErosion);
	
  // Print Success
  printf("<('.'<)\n");
  printf("(>'.')>\n");
  printf("\\(-_-)/\n");
  printf("/\\/\\(;;)/\\/\\\n");
  return 0;
}
